// Template for Assignment 1: CUDA
// Use "icc -O -openmp" to compile


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <sys/time.h>
#define threshold 1e-4
#define n (2048)
void init(void);
void ref(void);
void test(void);
void compare(int N, double *wref, double *w);
__global__ void test_kernel(int N, double *A, double *B, double *X);
void Transpose(int N, double M[n][n]);
double rtclock(void);

double a[n][n],b[n][n],x[n][n],xref[n][n];

int main(){

double clkbegin, clkend, t;

  printf("Matrix Size = %d\n",n);

  init();
  clkbegin = rtclock();
  ref();
  clkend = rtclock();
  t = clkend-clkbegin;
  printf("Mult-Tri-Solve-Seq: Approx GFLOPS: %.1f ; Time = %.3f sec; xref[n/2][n/2-1] = %f; \n",
1.0*n*n*n/t/1e9,t,xref[n/2][n/2-1]);

   clkbegin = rtclock();
   test();
   clkend = rtclock();
   t = clkend-clkbegin;
   printf("Multi-Tri-Solve-GPU: Approx GFLOPS: %.1f ; Time = %.3f sec; x[n/2][n/2-1] = %f; \n",
     1.0*n*n*n/t/1e9,t,x[n/2][n/2-1]);
   compare(n, (double *) x,(double *) xref);
}

__global__ void test_kernel(int N, double *A, double *B, double *X)
{
int i,j,k;
double temp;
// Template version uses only one thread, which does all the work
// This must be changed (and the launch parameters) to exploit GPU parallelism
// You can make any changes; only requirement is that correctness test passes
  k = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x+threadIdx.x);
  //if(threadIdx.x == 0) {
    //for(k=0;k<n;k++){
    /*
    if(k<n){
      for (i=0;i<n;i++){
        temp = B[k*N+i]; // temp = b[k][i];
        for (j=0;j<i;j++) temp = temp - A[i*N+j] * X[k*N+j]; // temp = temp - a[i][j]*x[k][j];
        X[k*N+i] = temp/A[i*N+i]; //x[k][i] = temp/a[i][i];
      }
    }
    */
    if(k<n){
      for (i=0;i<n;i++){
        temp = B[i*N+k]; // temp = b[k][i];
        for (j=0;j<i;j++) temp = temp - A[j*N+i] * X[j*N+k]; // temp = temp - a[i][j]*x[k][j];
        X[i*N+k] = temp/A[i*N+i]; //x[k][i] = temp/a[i][i];
      }
    }
//  }
// }
}

void test(void)
{
  double *Ad,*Bd,*Xd;
  int size;
  size = sizeof(double)*n*n;
  hipMalloc((void **) &Ad,size);
  hipMalloc((void **) &Bd,size);
  hipMalloc((void **) &Xd,size);
  Transpose(n,a);Transpose(n,b);
  hipMemcpy(Ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(Bd,b,size,hipMemcpyHostToDevice);
  dim3 dimGrid(32,32);
  dim3 dimBlock(2,2);
  test_kernel<<<dimGrid,dimBlock>>>(n,Ad,Bd,Xd);
  hipMemcpy(x,Xd,size,hipMemcpyDeviceToHost);
  Transpose(n,x);

}

void ref(void)
{
int i,j,k;
double temp;

  for(k=0;k<n;k++){
    for (i=0;i<n;i++)
    {
      temp = b[k][i];
      for (j=0;j<i;j++) temp = temp - a[i][j]*xref[k][j];
      xref[k][i] = temp/a[i][i];
    }
  }
}

void init(void)
{
int i,j,k;

  for(k=0;k<n;k++)
//    for(i=0;i<n;i++) { x[k][i] = k+i; a[k][i] = 1.0 + rand();}
    for(i=0;i<n;i++) { x[k][i] = k+i; a[k][i] = 1.0*(k+i+1)/(n+1);}
  for(k=0;k<n;k++)
    for(i=0;i<n;i++)
     { b[k][i]=0;
       for(j=0;j<=i;j++)
        b[k][i] += a[i][j]*x[k][j];
     }
  for(i=0;i<n;i++)
   for (j=0;j<n;j++)
   { x[i][j] = 0.0; xref[i][j] = 0.0; }
}

void compare(int N, double *wref, double *w)
{
double maxdiff,this_diff;
int numdiffs;
int i,j;
  numdiffs = 0;
  maxdiff = 0;
  for (i=0;i<N;i++)
   for (j=0;j<N;j++)
    {
     this_diff = wref[i*N+j]-w[i*N+j];
     if (this_diff < 0) this_diff = -1.0*this_diff;
     if (this_diff>threshold)
      { numdiffs++;
        if (this_diff > maxdiff) maxdiff=this_diff;
      }
    }
   if (numdiffs > 0)
      printf("%d Diffs found over threshold %f; Max Diff = %f\n",
               numdiffs,threshold,maxdiff);
   else
      printf("No differences found between reference and test versions\n");
}

void Transpose(int N, double M[n][n]) {
  int i,j;
  double temp;
  for(i=0;i<N;i++){
    for(j=0;j<i;j++){
      temp = M[i][j];
      M[i][j] = M[j][i];
      M[j][i] = temp;
    }
  }
}

double rtclock(void)
{
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday (&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d",stat);
  return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}
